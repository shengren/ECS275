#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

// output, ray generation
rtBuffer<float3, 2> subpixel_accumulator;
rtBuffer<float4, 2> output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(uint, sqrt_num_subpixels, , );

RT_PROGRAM void ot_ray_generation() {
  float3 result = make_float3(0.0f);
  for (int i = 0; i < sqrt_num_subpixels; ++i)
    for (int j = 0; j < sqrt_num_subpixels; ++j) {
      uint2 index = make_uint2(launch_index.x * sqrt_num_subpixels + i,
                               launch_index.y * sqrt_num_subpixels + j);
      result += subpixel_accumulator[index];
    }
  result /= (float)(sqrt_num_subpixels * sqrt_num_subpixels);
  result /= (float)frame_number;
  output_buffer[launch_index] = make_float4(result, 0.0f);
}
