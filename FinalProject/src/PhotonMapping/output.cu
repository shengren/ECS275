#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

// output, ray generation
rtBuffer<float3, 2> accumulator;
rtBuffer<float4, 2> output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(uint, sqrt_num_subpixels, , );

RT_PROGRAM void ot_ray_generation() {
  float3 result = accumulator[launch_index] / (float)frame_number;
  output_buffer[launch_index] = make_float4(result, 0.0f);
}
