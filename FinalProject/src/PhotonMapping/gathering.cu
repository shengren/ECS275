#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"
#include "kdtree.h"
#include "structs.h"
#include "inlines.h"

using namespace optix;

// variables used in multiple programs
//rtBuffer<float4, 2> output_buffer;
rtBuffer<float3, 2> subpixel_accumulator;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

// gathering, ray generation
rtBuffer<HitRecord, 2> hit_record_buffer;
rtBuffer<PhotonRecord, 1> photon_map;  // 1D
rtBuffer<ParallelogramLight> lights;  // to-do: only have parallelogram lights
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint, gt_shadow_ray_type, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float, radius2, , );

// to-do: photon_map (rtBuffer) cannot be passed through function parameters
// hence this function is not included in 'inlines.h'
// output_buffer is used as a debug buffer in some places
// modified from gather() in progressivePhotonMap/ppm_gather.cu
#define MAX_DEPTH 20  // one MILLION photons

__device__ __inline__ void estimateRadiance(const HitRecord& hr,
                                            const float& radius2,
                                            float3& total_flux,
                                            int& num_photons) {
  total_flux = make_float3(0.0f, 0.0f, 0.0f);
  num_photons = 0;  // to-do: unused now

  unsigned int stack[MAX_DEPTH];
  unsigned int stack_current = 0;
  unsigned int node = 0;  // 0 is the start

#define push_node(N) stack[stack_current++] = (N)
#define pop_node()   stack[--stack_current]

  push_node(0);

  int photon_map_size = photon_map.size();  // for debugging

  do {
    // debugging assertion
    if (!(node < photon_map_size)) {
      //output_buffer[launch_index] = make_float4(1.0f, 1.0f, 0.0f, 0.0f);
      subpixel_accumulator[launch_index] = make_float3(1.0f, 1.0f, 0.0f);
      return;
    }

    const PhotonRecord& pr = photon_map[node];

    if (!(pr.axis & PPM_NULL)) {
      float3 diff = hr.position - pr.position;
      float distance2 = dot(diff, diff);

      // accumulate photons
      if (distance2 <= radius2) {
        if (dot(hr.normal, pr.normal) > 1e-2f) {  // on the same plane?
          total_flux += pr.power * getDiffuseBRDF(hr.Rho_d);  // with BRDF
          num_photons++;
        }
      }

      // Recurse
      if (!(pr.axis & PPM_LEAF)) {
        float d;
        if (pr.axis & PPM_X) d = diff.x;
        else if (pr.axis & PPM_Y) d = diff.y;
        else d = diff.z;  // PPM_Z

        // Calculate the next child selector. 0 is left, 1 is right.
        int selector = d < 0.0f ? 0 : 1;
        if (d * d < radius2) {
          // debugging assertion
          if (!(stack_current + 1 < MAX_DEPTH)) {
            //output_buffer[launch_index] = make_float4(1.0f, 1.0f, 0.0f, 0.0f);
            subpixel_accumulator[launch_index] = make_float3(1.0f, 1.0f, 0.0f);
            return;
          }

          push_node((node << 1) + 2 - selector);
        }

        // debugging assertion
        if (!(stack_current + 1 < MAX_DEPTH)) {
          //output_buffer[launch_index] = make_float4(1.0f, 1.0f, 0.0f, 0.0f);
          subpixel_accumulator[launch_index] = make_float3(1.0f, 1.0f, 0.0f);
          return;
        }

        node = (node << 1) + 1 + selector;
      } else {
        node = pop_node();
      }
    } else {
      node = pop_node();
    }
  } while (node);
}

// to-do: make this function separate in order to make the code clean
// launch_index, launch_dim, frame_number, lights are local variables
__device__ __inline__ float3 directIllumination(const HitRecord& hr) {
  uint seed = tea<16>(launch_index.y * launch_dim.x + launch_index.x,
                      frame_number);

  // uniformly choose one of the area lights
  int i = (int)((float)lights.size() * rnd(seed));
  const ParallelogramLight& light = lights[i];  // to-do: only one type of light now

  float3 jitter_scale_v1 = light.v1 / (float)light.sqrt_num_samples;
  float3 jitter_scale_v2 = light.v2 / (float)light.sqrt_num_samples;
  int num_samples = light.sqrt_num_samples * light.sqrt_num_samples;
  float ratio = 0.0;
  for (int x = 0; x < light.sqrt_num_samples; ++x)
    for (int y = 0; y < light.sqrt_num_samples; ++y) {
      float3 sample_on_light = light.corner +
                               jitter_scale_v1 * (x + rnd(seed)) +
                               jitter_scale_v2 * (y + rnd(seed));
      float distance_to_light = length(sample_on_light - hr.position);
      float3 direction_to_light = normalize(sample_on_light - hr.position);

      if (dot(hr.normal, direction_to_light) > 1e-5f &&
          dot(light.normal, -direction_to_light) > 1e-5f) {  // trace shadow ray
        GTShadowRayPayload payload;
        payload.blocked = false;

        Ray ray(hr.position,
                direction_to_light,
                gt_shadow_ray_type,
                1e-10f,
                distance_to_light - 1e-10f);

        rtTrace(top_object, ray, payload);  // to-do: hitting the light source doesn't count

        if (!payload.blocked) {
          float geom = getGeometry(hr.normal,
                                   light.normal,
                                   direction_to_light,
                                   distance_to_light);
          ratio += geom;
        }
      }
    }
  ratio *= light.area;  // probability of sampling on this light source
  ratio *= (float)lights.size();  // probability of sampling among light sources
  ratio /= (float)num_samples;

  return light.emitted * getDiffuseBRDF(hr.Rho_d) * ratio;
}

RT_PROGRAM void gt_ray_generation() {
  // clean the output buffer
  if (frame_number == 1) {
    //output_buffer[launch_index] = make_float4(0.0f);
    subpixel_accumulator[launch_index] = make_float3(0.0f);
  }

  const HitRecord& hr = hit_record_buffer[launch_index];

  if (!(hr.flags & HIT)) {
    //output_buffer[launch_index] = make_float4(hr.attenuation);
    subpixel_accumulator[launch_index] += hr.attenuation;
    return;
  }

  // indirect illumination
  float3 total_flux = make_float3(0.0f);
  int num_photons = 0;  // to-do: unused now

  estimateRadiance(hr, radius2, total_flux, num_photons);

  float3 indirect = total_flux / (M_PI * radius2);

  // direct illumination
  float3 direct = directIllumination(hr);

  // output
  //float3 result = direct * hr.attenuation;
  //float3 result = indirect * hr.attenuation;
  float3 result = (direct + indirect) * hr.attenuation;

  /*
  if (frame_number == 1) {
    output_buffer[launch_index] = make_float4(result, 0.0f);
  } else {
    float a = 1.0f / (float)frame_number;
    float b = ((float)frame_number - 1.0f) * a;
    float3 old_result = make_float3(output_buffer[launch_index]);
    output_buffer[launch_index] = make_float4(a * result + b * old_result, 0.0f);
  }
  */
  subpixel_accumulator[launch_index] += result;
}

// gathering, exception
rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void gt_exception() {
  //output_buffer[launch_index] = make_float4(bad_color);
  subpixel_accumulator[launch_index] = bad_color;

  rtPrintExceptionDetails();  // to-do: for debugging
}

// gathering, direct illumination, shadow ray, any hit
rtDeclareVariable(GTShadowRayPayload, gt_shadow_ray_payload, rtPayload, );

RT_PROGRAM void gt_shadow_ray_any_hit() {
  gt_shadow_ray_payload.blocked = true;
  rtTerminateRay();  // to-do: what's the difference between this function and 'return'
}
