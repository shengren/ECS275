#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"
#include "kdtree.h"
#include "structs.h"
#include "inlines.h"

using namespace optix;

// gathering, ray generation
rtBuffer<HitRecord, 2> hit_record_buffer;
rtBuffer<float4, 2> output_buffer;
rtBuffer<PhotonRecord, 1> photon_map;  // 1D
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(float, total_emitted, , );
rtDeclareVariable(uint, gt_shadow_ray_type, , );
rtBuffer<ParallelogramLight> lights;  // to-do: only have parallelogram lights
rtDeclareVariable(rtObject, top_object, , );

// to-do:
#define MAX_DEPTH 20

RT_PROGRAM void gt_ray_generation() {
  HitRecord hr = hit_record_buffer[launch_index];

  if ((hr.flags & EXCEPTION) || !(hr.flags & HIT)) {
    output_buffer[launch_index] = make_float4(hr.attenuated_Kd);
    return;
  }

  float3 total_flux = make_float3(0.0f);
  int num_photons = 0;
  //float radius2 = 0.25f;  // to-do: input parameter!!!
  float radius2 = 5.0f;  // to-do: input parameter!!!

  // modified from gather() in ppm_gather.cu
  // begin
  unsigned int stack[MAX_DEPTH];
  unsigned int stack_current = 0;
  unsigned int node = 0;

#define push_node(N) stack[stack_current++] = (N)
#define pop_node()   stack[--stack_current]

  push_node(0);

  int photon_map_size = photon_map.size();
  if (launch_index.x < 2 && launch_index.y < 2) {
    output_buffer[launch_index] = make_float4((float)photon_map_size, 0.0, 0.0, 0.0);
    return;
  }

  do {
    // check
    if (!(node < photon_map_size)) {
      output_buffer[launch_index] = make_float4(1.0, 1.0, 0.0, 0.0);
      return;
    }

    PhotonRecord& pr = photon_map[node];

    if (!(pr.axis & PPM_NULL)) {
      float3 diff = hr.position - pr.position;
      float distance2 = dot(diff, diff);

      // accumulate photon
      if (distance2 <= radius2) {
        /*
        if (dot(pr.normal, hr.normal) > 1e-5) {
          //total_flux += pr.power * hr.attenuated_Kd;
          total_flux += pr.power;
          num_photons++;
        }
        */
        total_flux += pr.power;
        num_photons++;
      }

      // Recurse
      if (!(pr.axis & PPM_LEAF)) {
        float d;
        if (pr.axis & PPM_X) d = diff.x;
        else if (pr.axis & PPM_Y) d = diff.y;
        else d = diff.z;  // PPM_Z

        // Calculate the next child selector. 0 is left, 1 is right.
        int selector = d < 0.0f ? 0 : 1;
        if (d * d < radius2) {
          // check
          if (!(stack_current + 1 < MAX_DEPTH)) {
            output_buffer[launch_index] = make_float4(0.0, 1.0, 0.0, 0.0);
            return;
          }

          push_node((node << 1) + 2 - selector);
        }

        // check
        if (!(stack_current + 1 < MAX_DEPTH)) {
          output_buffer[launch_index] = make_float4(0.0, 1.0, 1.0, 0.0);
          return;
        }

        node = (node << 1) + 1 + selector;
      } else {
        node = pop_node();
      }
    } else {
      node = pop_node();
    }
  } while (node);
  // end

  // indirect
  // to-do:
  //float3 indirect = total_flux / (M_PI * radius2) / total_emitted;
  float3 indirect;
  if (num_photons > 0)
    indirect = make_float3(1.0f);
  else
    indirect = make_float3(0.0f);

  // direct
  float3 direct = make_float3(0.0f);
  /*
  uint seed = tea<16>(launch_index.y * launch_dim.x + launch_index.x,
                      frame_number);
  // uniformly choose one of the area lights
  int i = (int)((float)lights.size() * rnd(seed));
  ParallelogramLight light = lights[i];  // to-do: only one type of light now

  float3 jitter_scale_v1 = light.v1 / (float)light.sqrt_num_samples;
  float3 jitter_scale_v2 = light.v2 / (float)light.sqrt_num_samples;
  int num_samples = light.sqrt_num_samples * light.sqrt_num_samples;
  float ratio = 0.0;
  for (int x = 0; x < light.sqrt_num_samples; ++x)
    for (int y = 0; y < light.sqrt_num_samples; ++y) {
      float3 sample_on_light =
          light.corner +
          jitter_scale_v1 * (x + rnd(seed)) +
          jitter_scale_v2 * (y + rnd(seed));
      float distance_to_light = length(sample_on_light - hr.position);
      float3 direction_to_light = normalize(sample_on_light - hr.position);

      if (dot(hr.normal, direction_to_light) > 0.0f &&
          dot(light.normal, -direction_to_light) > 0.0f) {  // trace shadow ray
        GTShadowRayPayload payload;
        payload.blocked = false;

        Ray ray(hr.position,
                direction_to_light,
                gt_shadow_ray_type,
                1e-10f,
                distance_to_light - 1e-10f);

        rtTrace(top_object, ray, payload);  // to-do: hitting the light source doesn't count

        if (!payload.blocked) {
          float BRDF = getDiffuseBRDF();
          float geom = getGeometry(hr.normal,
                                   light.normal,
                                   direction_to_light,
                                   distance_to_light);
          ratio += BRDF * geom;
        }
      }
    }
  ratio *= light.area / (float)num_samples;
  ratio *= (float)lights.size();
  direct = light.emitted * ratio;
  */

  // output
  //output_buffer[launch_index] = make_float4((indirect + direct) * hr.attenuated_Kd,
  //                                          0.0f);
  output_buffer[launch_index] = make_float4((indirect + direct) * hr.Rho_d * hr.attenuated_Kd,
                                            0.0f);
}

// gathering, exception
rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void gt_exception() {
  output_buffer[launch_index] = make_float4(bad_color);
}

// gathering, direct illumination, shadow ray, any hit
rtDeclareVariable(GTShadowRayPayload, gt_shadow_ray_payload, rtPayload, );

RT_PROGRAM void gt_shadow_ray_any_hit() {
  gt_shadow_ray_payload.blocked = true;
  rtTerminateRay();
}
