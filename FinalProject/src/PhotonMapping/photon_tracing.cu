#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"  // tea<>, rnd
#include "structs.h"
#include "inlines.h"

using namespace optix;

// modified from progressivePhotonMap/path_tracer.h
__device__ __inline__ optix::float3 sampleUnitHemisphereCosine(
    optix::uint& seed,
    const optix::float3& normal) {
  optix::float3 U, V, W;
  createONB(normal, U, V, W);

  float phi = 2.0f * M_PIf * rnd(seed);
  float r = sqrt(rnd(seed));
  float x = r * cos(phi);
  float y = r * sin(phi);
  float z = 1.0f - x * x - y * y;
  z = z > 0.0f ? sqrt(z) : 0.0f;

  return x * U + y * V + z * W;
}

__device__ __inline__ void generatePhoton(const ParallelogramLight& light,
                                          uint& seed,
                                          float3& sample_position,
                                          float3& sample_direction,
                                          float3& sample_power) {
  sample_position = light.corner + light.v1 * rnd(seed) + light.v2 * rnd(seed);

  sample_direction = sampleUnitHemisphereCosine(seed, light.normal);

  sample_power = light.power;
}

// variables used in multiple programs
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, pt_photon_ray_type, , );
rtDeclareVariable(uint, max_num_deposits, , );
rtBuffer<PhotonRecord> photon_record_buffer;  // 1D by default

// photon tracing, ray generation
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint, frame_number, , );
rtBuffer<ParallelogramLight> lights;  // to-do: only have parallelogram lights

RT_PROGRAM void pt_ray_generation() {
  uint index = launch_index.y * launch_dim.x + launch_index.x;  // to-do: Will 1D launch be enough?
  uint seed = tea<16>(index, frame_number);
  float3 sample_position;
  float3 sample_direction;
  float3 sample_power;

  // to-do: better way?
  for (int i = 0; i < max_num_deposits; ++i)
    photon_record_buffer[index + i].power = make_float3(0.0f);

  // to-do: one parallelogram light only
  generatePhoton(lights[0], seed, sample_position, sample_direction, sample_power);

  Ray ray(sample_position,
          sample_direction,
          pt_photon_ray_type,
          1e-10f);

  PTPhotonRayPayload payload;
  payload.power = sample_power;
  payload.index = index;
  payload.seed = seed;
  payload.num_deposits = 0;
  payload.depth = 0;

  rtTrace(top_object, ray, payload);
}

// photon tracing, exception
RT_PROGRAM void pt_exception() {
  rtPrintExceptionDetails();  // to-do:
}

// photon tracing, photon ray, closest hit, default material
rtDeclareVariable(Ray, pt_photon_ray, rtCurrentRay, );
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(PTPhotonRayPayload, pt_photon_ray_payload, rtPayload, );
rtDeclareVariable(uint, max_depth, , );

RT_PROGRAM void pt_photon_ray_closest_hit() {
  if (fmaxf(Le) > 0.0f) {  // light source
    return;
  }

  float3 hit_point = pt_photon_ray.origin + hit_t * pt_photon_ray.direction;
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -pt_photon_ray.direction, world_geometric_normal);

  // record when hit diffuse surfaces and bounced at least once (avoid doubling direct illumination)
  // to-do: test-only
  //if (fmaxf(Kd) > 0.0f && pt_photon_ray_payload.depth > 0) {
  if (fmaxf(Kd) > 0.0f) {
    PhotonRecord& pr = photon_record_buffer[pt_photon_ray_payload.index +
                                            pt_photon_ray_payload.num_deposits];
    pr.position = hit_point;
    pr.normal = ffnormal;
    pr.incoming = -pt_photon_ray.direction;  // hit_point is the origin
    pr.power = pt_photon_ray_payload.power;

    pt_photon_ray_payload.num_deposits++;
  }

  if (pt_photon_ray_payload.num_deposits >= max_num_deposits ||
      pt_photon_ray_payload.depth >= max_depth) {
    return;
  }

  pt_photon_ray_payload.depth++;
  float3 next_direction;
  if (fmaxf(Kd) > 0.0f) {  // diffuse
    pt_photon_ray_payload.power *= Kd;  // to-do: BRDF shouldn't be a constant
    next_direction = sampleUnitHemisphereCosine(pt_photon_ray_payload.seed,
                                                ffnormal);
  } else {  // specular
    pt_photon_ray_payload.power *= Ks;  // to-do: BRDF shouldn't be a constant
    next_direction = reflect(pt_photon_ray.direction, ffnormal);  // inversed incoming
  }

  Ray ray(hit_point,
          next_direction,
          pt_photon_ray_type,
          1e-10f);
  rtTrace(top_object, ray, pt_photon_ray_payload);
}

// photon tracing, photon, miss, default material
// do nothing
