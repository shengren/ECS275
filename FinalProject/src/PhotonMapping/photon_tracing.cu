#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"  // tea<>, rnd
#include "structs.h"
#include "inlines.h"

using namespace optix;

// variables used in multiple programs
rtBuffer<PhotonRecord, 1> photon_record_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, pt_photon_ray_type, , );
rtDeclareVariable(uint, max_num_deposits, , );

// photon tracing, ray generation
rtBuffer<ParallelogramLight> lights;  // to-do: only have parallelogram lights
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(float, total_emitted, , );

RT_PROGRAM void pt_ray_generation() {
  uint index = launch_index.y * launch_dim.x + launch_index.x;  // to-do: is 1D launch enough?
  uint seed = tea<16>(index, frame_number);
  float3 sample_position;
  float3 sample_direction;
  float3 sample_power;

  // to-do: better way?
  // initialize photon records assigned to this photon
  for (int i = 0; i < max_num_deposits; ++i) {
    photon_record_buffer[index * max_num_deposits + i].power = 
    photon_record_buffer[index * max_num_deposits + i].position = 
    photon_record_buffer[index * max_num_deposits + i].normal = 
    photon_record_buffer[index * max_num_deposits + i].incoming = 
      make_float3(0.0f);
    photon_record_buffer[index * max_num_deposits + i].axis = 0;
  }

  // to-do: only one parallelogram light now
  generatePhoton(lights[0], seed, sample_position, sample_direction, sample_power);

  Ray ray(sample_position,
          sample_direction,
          pt_photon_ray_type,
          1e-2f);

  PTPhotonRayPayload payload;
  payload.power = sample_power / total_emitted;  // to-do: real power per photon?
  payload.index = index;
  payload.num_deposits = 0;
  payload.depth = 1;
  payload.seed = seed;
  payload.inside = false;

  rtTrace(top_object, ray, payload);
}

// photon tracing, exception
RT_PROGRAM void pt_exception() {
  rtPrintExceptionDetails();  // to-do: for debugging
}

// photon tracing, photon ray, closest hit, default material
rtDeclareVariable(Ray, pt_photon_ray, rtCurrentRay, );
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(PTPhotonRayPayload, pt_photon_ray_payload, rtPayload, );
rtDeclareVariable(uint, min_depth, , );  // started from 1, record bounces in [min_depth, max_depth]
rtDeclareVariable(uint, max_depth, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(float3, Rho_d, , );
rtDeclareVariable(float3, Rho_s, , );
rtDeclareVariable(float, shininess, , );  // unused now
rtDeclareVariable(float, index_of_refraction, , );  // non-zero indiates a refraction surface, Rho_s is needed as well

RT_PROGRAM void pt_photon_ray_closest_hit() {
  if (fmaxf(Le) > 0.0f) {  // light source
    return;
  }

  float3 hit_point = pt_photon_ray.origin + hit_t * pt_photon_ray.direction;
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -pt_photon_ray.direction, world_geometric_normal);

  // record when hit diffuse surfaces and bounced at least once (avoid doubling direct illumination)
  // min_depth = 1, record from the first bounce for test, = 2, regular case
  if (fmaxf(Rho_d) > 0.0f && pt_photon_ray_payload.depth >= min_depth) {
    PhotonRecord& pr = photon_record_buffer[pt_photon_ray_payload.index * max_num_deposits +
                                            pt_photon_ray_payload.num_deposits];
    pr.power = pt_photon_ray_payload.power;
    pr.position = hit_point;
    pr.normal = ffnormal;
    pr.incoming = -pt_photon_ray.direction;  // hit_point is the origin
    // pr.axis is used in kdtree

    pt_photon_ray_payload.num_deposits++;
  }

  if (pt_photon_ray_payload.num_deposits >= max_num_deposits ||
      pt_photon_ray_payload.depth >= max_depth) {
    return;
  }

  float3 next_direction;
  // to-do: the material is either pure diffuse or pure specular, no third option now
  if (fmaxf(Rho_d) > 0.0f) {  // diffuse
    next_direction = sampleUnitHemisphereCosine(pt_photon_ray_payload.seed,
                                                ffnormal);
    pt_photon_ray_payload.power *= getDiffuseBRDF(Rho_d);
  } else if (index_of_refraction > 0.0) {  // refraction
    float iof = (pt_photon_ray_payload.inside) ?
                (1.0f / index_of_refraction) : index_of_refraction;
    refract(next_direction, pt_photon_ray.direction, ffnormal, iof);
    if (pt_photon_ray_payload.inside) {
      //float p = max(hit_t, 1.0f);
      //pt_photon_ray_payload.power *= powf(Rho_s.x, p);  // Beer's law, assume Rho_x=y=z
      pt_photon_ray_payload.power *= Rho_s;
    }
    pt_photon_ray_payload.inside = !pt_photon_ray_payload.inside;
  } else {  // perfect specular
    next_direction = reflect(pt_photon_ray.direction, ffnormal);  // inversed incoming
    //pt_photon_ray_payload.power *= getSpecularBRDF(-pt_photon_ray.direction,  // incoming
    //                                               ffnormal,
    //                                               next_direction,  // outgoing
    //                                               Rho_s,
    //                                               shininess);
    pt_photon_ray_payload.power *= Rho_s;
  }
  // to-do: confirm that there is no cosine term during photon transport
  //pt_photon_ray_payload.power *= dot(-pt_photon_ray.direction, ffnormal);  // cosine term
  pt_photon_ray_payload.depth++;

  Ray ray(hit_point,
          next_direction,
          pt_photon_ray_type,
          1e-2f);

  rtTrace(top_object, ray, pt_photon_ray_payload);
}

// photon tracing, photon, miss, default material
// do nothing
