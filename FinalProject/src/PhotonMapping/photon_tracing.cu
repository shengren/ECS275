#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"  // tea<>, rnd
#include "structs.h"
#include "inlines.h"

using namespace optix;

// variables used in multiple programs
rtBuffer<PhotonRecord, 1> photon_record_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, pt_photon_ray_type, , );
rtDeclareVariable(uint, max_num_deposits, , );

// photon tracing, ray generation
rtBuffer<ParallelogramLight> lights;  // to-do: only have parallelogram lights
//rtBuffer<Sphere> caustics;  // to-do: only caustic spheres
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(float, total_emitted, , );

// since we reuse the photon_record_buffer, make sure it is clean before generating photons
// now it is cleaned before executing this program.
RT_PROGRAM void pt_ray_generation() {
  uint index = launch_index.y * launch_dim.x + launch_index.x;  // to-do: is 1D launch enough?
  uint seed = tea<16>(index, frame_number);
  float3 sample_position;
  float3 sample_direction;
  float3 sample_power;

  generatePhoton(lights[0], seed,
      sample_position, sample_direction, sample_power);  // to-do: only one parallelogram light now

  Ray ray(sample_position,
          sample_direction,
          pt_photon_ray_type,
          1e-2f);

  PTPhotonRayPayload payload;
  payload.power = sample_power / total_emitted;  // to-do: real power per photon?
  payload.index = index;
  payload.num_deposits = 0;
  payload.depth = 1;
  payload.seed = seed;
  payload.inside = false;

  rtTrace(top_object, ray, payload);
}

// photon tracing, exception
RT_PROGRAM void pt_exception() {
  rtPrintExceptionDetails();  // to-do: for debugging
}

// photon tracing, photon ray, closest hit, default material
rtDeclareVariable(Ray, pt_photon_ray, rtCurrentRay, );
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(PTPhotonRayPayload, pt_photon_ray_payload, rtPayload, );
rtDeclareVariable(uint, min_depth, , );  // started from 1, record bounces in [min_depth, max_depth]
rtDeclareVariable(uint, max_depth, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(float3, Rho_d, , );
rtDeclareVariable(float3, Rho_s, , );
rtDeclareVariable(float, shininess, , );  // unused now
rtDeclareVariable(float, index_of_refraction, , );  // non-zero indiates a refraction surface, Rho_s is needed as well

RT_PROGRAM void pt_photon_ray_closest_hit() {
  if (pt_photon_ray_payload.num_deposits >= max_num_deposits ||
      pt_photon_ray_payload.depth > max_depth) {
    return;
  }

  if (fmaxf(Le) > 0.0f) {  // light source
    return;
  }

  float3 hit_point = pt_photon_ray.origin + hit_t * pt_photon_ray.direction;
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -pt_photon_ray.direction, world_geometric_normal);

  // record when hit diffuse surfaces and bounced at least once (avoid doubling direct illumination)
  // min_depth = 1, record from the first bounce for test, = 2, regular case
  if (fmaxf(Rho_d) > 0.0f && pt_photon_ray_payload.depth >= min_depth) {
    PhotonRecord& pr = photon_record_buffer[pt_photon_ray_payload.index +
                                            pt_photon_ray_payload.num_deposits];
    pr.power = pt_photon_ray_payload.power;
    pr.position = hit_point;
    pr.normal = ffnormal;
    pr.incoming = -pt_photon_ray.direction;  // hit_point is the origin
    // pr.axis is used in kdtree

    pt_photon_ray_payload.num_deposits++;
  }

  pt_photon_ray_payload.depth++;

  if (fmaxf(Rho_d) > 0.0f) {  // diffuse
    float3 next_direction = sampleUnitHemisphereCosine(pt_photon_ray_payload.seed,
                                                       ffnormal);
    pt_photon_ray_payload.power *= getDiffuseBRDF(Rho_d);
    Ray ray(hit_point,
            next_direction,
            pt_photon_ray_type,
            1e-2f);
    rtTrace(top_object, ray, pt_photon_ray_payload);
    return;
  }

  // specular

  float3 reflection_direction = reflect(pt_photon_ray.direction, ffnormal);  // inversed incoming
  float reflection_ratio = 1.0f;
  float3 refraction_direction;
  float refraction_ratio = 0.0f;
  bool has_refraction = false;
  PTPhotonRayPayload refraction_payload = pt_photon_ray_payload;
  if (index_of_refraction > 0.0f) {
    float iof = (pt_photon_ray_payload.inside) ?
                (1.0f / index_of_refraction) : index_of_refraction;
    refract(refraction_direction, pt_photon_ray.direction, ffnormal, iof);
    float cos_i = dot(-pt_photon_ray.direction, ffnormal);
    float cos2_t = 1.0f - ((1.0f - (cos_i * cos_i)) / (iof * iof));
    if (cos2_t >= 0) {
      has_refraction = true;
      float a = index_of_refraction - 1.0f;
      float b = index_of_refraction + 1.0f;
      float R0 = a * a / (b * b);
      float c = 1.0f - (pt_photon_ray_payload.inside ?
                        dot(refraction_direction, -ffnormal) :
                        cos_i);
      reflection_ratio = R0 + (1.0f - R0) * c * c * c * c * c;
      refraction_ratio = 1.0f - reflection_ratio;
    }
  }

  // refraction
  if (has_refraction) {
    refraction_payload.power *= Rho_s * refraction_ratio;
    refraction_payload.inside = !refraction_payload.inside;
    Ray ray(hit_point,
            refraction_direction,
            pt_photon_ray_type,
            1e-2f);
    rtTrace(top_object, ray, refraction_payload);
    // update num_deposits is important. Otherwise, the photons generated by
    // refraction will be overwritten.
    pt_photon_ray_payload.num_deposits = refraction_payload.num_deposits;
    pt_photon_ray_payload.seed = refraction_payload.seed;
  }

  // reflection
  // to-do: no internal reflection
  if (pt_photon_ray_payload.inside)
    return;

  pt_photon_ray_payload.power *= Rho_s * reflection_ratio;  // perfect reflection
  Ray ray(hit_point,
          reflection_direction,
          pt_photon_ray_type,
          1e-2f);
  rtTrace(top_object, ray, pt_photon_ray_payload);
}

// photon tracing, photon, miss, default material
// do nothing
