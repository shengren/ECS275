#include "hip/hip_runtime.h"
#include <optix.h>  // must include these two headers to compile
#include <optixu/optixu_math_namespace.h>

#include "hip/device_functions.h"

#include <cstdio>

void TestDeviceFunction(void) {
  printf("before calling the test kernel.\n");
  
  int num_elements = 256;
  int num_bytes = num_elements * sizeof(int);

  int *host_array = (int*)malloc(num_bytes);
  int *device_array = 0;
  hipMalloc((void**)&device_array, num_bytes);

  int block_size = 128;
  int num_blocks = num_elements / block_size;

  device_test<<<num_blocks, block_size>>>(device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  for (int i = 0; i < num_elements; ++i)
    printf("%d ", host_array[i]);
  printf("\n");

  free(host_array);
  hipFree(device_array);

  printf("after calling the test kernel.\n");
}

__global__ void device_test(int *array) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  array[index] = 255 - index;
}
