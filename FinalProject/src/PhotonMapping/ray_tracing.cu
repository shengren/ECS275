#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"  // tea<>, rnd()
#include "structs.h"
#include "inlines.h"

using namespace optix;

// variables used in multiple programs
rtBuffer<HitRecord, 2> hit_record_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, rt_viewing_ray_type, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(RTViewingRayPayload, rt_viewing_ray_payload, rtPayload, );

// ray tracing, ray generation
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(float3, camera_position, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float3, camera_w, , );

RT_PROGRAM void rt_ray_generation() {
  uint seed = tea<16>(launch_index.y * launch_dim.x + launch_index.x,
                      frame_number);
  float2 offset = (make_float2(launch_index)
                  + make_float2(rnd(seed), rnd(seed)))
                  / make_float2(launch_dim) * 2.0f - 1.0f;

  Ray ray(camera_position,  // origin
          normalize(offset.x * camera_u + offset.y * camera_v + camera_w),  // direction
          rt_viewing_ray_type,  // type
          1e-10f);  // tmin; tmax uses default

  RTViewingRayPayload payload;
  payload.attenuation = make_float3(1.0f);
  payload.depth = 1;  // to-do: unused now

  rtTrace(top_object, ray, payload);
}

// ray tracing, exception
rtDeclareVariable(float3, bad_color, , );  // green

RT_PROGRAM void rt_exception() {
  HitRecord& hr = hit_record_buffer[launch_index];
  hr.flags = EXCEPTION;
  hr.attenuation = bad_color;
  hr.position = hr.normal = hr.outgoing = hr.Rho_d = make_float3(0.0f);

  rtPrintExceptionDetails();  // to-do: for debugging
}

// ray tracing, viewing ray, closest hit, default material
rtDeclareVariable(Ray, rt_viewing_ray, rtCurrentRay, );
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(float3, Rho_d, , );
rtDeclareVariable(float3, Rho_s, , );
rtDeclareVariable(float, shininess, , );

RT_PROGRAM void rt_viewing_ray_closest_hit() {
  if (fmaxf(Le) > 0.0f) {  // light source?
    HitRecord& hr = hit_record_buffer[launch_index];
    hr.flags = HIT_LIGHT;
    hr.attenuation = rt_viewing_ray_payload.attenuation * Le;
    hr.position = hr.normal = hr.outgoing = hr.Rho_d = make_float3(0.0f);
    return;
  }

  float3 hit_point = rt_viewing_ray.origin + hit_t * rt_viewing_ray.direction;
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -rt_viewing_ray.direction, world_geometric_normal);

  if (fmaxf(Rho_d) > 0.0f) {  // diffuse surface?
    HitRecord& hr = hit_record_buffer[launch_index];
    hr.flags = HIT;
    // since we don't know the incoming directions, here we don't apply the
    // BRDF and cosine term. They should be computed in the gathering pass.
    // i.e. attenuation only includes all previous hits' computations on
    // specular surfaces.
    hr.attenuation = rt_viewing_ray_payload.attenuation;
    hr.position = hit_point;
    hr.normal = ffnormal;
    hr.outgoing = -rt_viewing_ray.direction;
    hr.Rho_d = Rho_d;
    return;
  }

  // specular surface, recursion
  rt_viewing_ray_payload.depth++;  // to-do: unused now
  float3 reflection_direction = reflect(rt_viewing_ray.direction, ffnormal);  // inversed incoming
  rt_viewing_ray_payload.attenuation *= getSpecularBRDF(reflection_direction,  // incoming
                                                        ffnormal,  // normal
                                                        -rt_viewing_ray.direction,  // outgoing
                                                        Rho_s,  // not Ks but for computing Ks
                                                        shininess);  // the power factor
  rt_viewing_ray_payload.attenuation *= dot(reflection_direction, ffnormal);  // cosine term
  Ray ray(hit_point,
          reflection_direction,
          rt_viewing_ray_type,
          1e-10f);
  rtTrace(top_object, ray, rt_viewing_ray_payload);
}

// ray tracing, viewing ray, miss, default material
rtDeclareVariable(float3, bg_color, , );  // black

RT_PROGRAM void rt_viewing_ray_miss() {
  HitRecord& hr = hit_record_buffer[launch_index];
  hr.flags = HIT_BACKGROUND;
  hr.attenuation = rt_viewing_ray_payload.attenuation * bg_color;
  hr.position = hr.normal = hr.outgoing = hr.Rho_d = make_float3(0.0f);
}
