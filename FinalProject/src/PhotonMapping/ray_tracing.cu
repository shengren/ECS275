#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"  // tea<>, rnd()
#include "structs.h"
#include "inlines.h"

using namespace optix;

// variables used in multiple programs
rtBuffer<HitRecord, 2> hit_record_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, rt_viewing_ray_type, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(RTViewingRayPayload, rt_viewing_ray_payload, rtPayload, );

// ray tracing, ray generation
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(float3, camera_position, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float3, camera_w, , );

RT_PROGRAM void rt_ray_generation() {
  uint seed = tea<16>(launch_index.y * launch_dim.x + launch_index.x,
                      frame_number);
  float2 offset = (make_float2(launch_index)
                  + make_float2(rnd(seed), rnd(seed)))
                  / make_float2(launch_dim) * 2.0f - 1.0f;

  Ray ray(camera_position,  // origin
          normalize(offset.x * camera_u + offset.y * camera_v + camera_w),  // direction
          rt_viewing_ray_type,  // type
          1e-2f);  // tmin; tmax uses default

  RTViewingRayPayload payload;
  payload.attenuation = make_float3(1.0f);
  payload.depth = 1;  // to-do: unused now
  payload.inside = false;

  rtTrace(top_object, ray, payload);
}

// ray tracing, exception
rtDeclareVariable(float3, bad_color, , );  // green

RT_PROGRAM void rt_exception() {
  HitRecord& hr = hit_record_buffer[launch_index];
  hr.flags = EXCEPTION;
  hr.attenuation = bad_color;
  hr.position = hr.normal = hr.outgoing = hr.Rho_d = make_float3(0.0f);

  rtPrintExceptionDetails();  // to-do: for debugging
}

// ray tracing, viewing ray, closest hit, default material
rtDeclareVariable(Ray, rt_viewing_ray, rtCurrentRay, );
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(float3, Rho_d, , );
rtDeclareVariable(float3, Rho_s, , );
rtDeclareVariable(float, shininess, , );  // unused now
rtDeclareVariable(float, index_of_refraction, , );  // non-zero indiates a refraction surface, Rho_s is needed as well

RT_PROGRAM void rt_viewing_ray_closest_hit() {
  if (fmaxf(Le) > 0.0f) {  // light source?
    HitRecord& hr = hit_record_buffer[launch_index];
    hr.flags = HIT_LIGHT;
    hr.attenuation = rt_viewing_ray_payload.attenuation * Le;
    hr.position = hr.normal = hr.outgoing = hr.Rho_d = make_float3(0.0f);
    return;
  }

  float3 hit_point = rt_viewing_ray.origin + hit_t * rt_viewing_ray.direction;
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -rt_viewing_ray.direction, world_geometric_normal);

  if (fmaxf(Rho_d) > 0.0f) {  // diffuse surface?
    HitRecord& hr = hit_record_buffer[launch_index];
    hr.flags = HIT;
    // since we don't know the incoming directions, here we don't apply the
    // BRDF and cosine term. They should be computed in the gathering pass.
    // i.e. attenuation only includes all previous hits' computations on
    // specular surfaces.
    hr.attenuation = rt_viewing_ray_payload.attenuation;
    hr.position = hit_point;
    hr.normal = ffnormal;
    hr.outgoing = -rt_viewing_ray.direction;
    hr.Rho_d = Rho_d;
    return;
  }

  rt_viewing_ray_payload.depth++;  // to-do: unused now
  float3 next_direction;
  if (index_of_refraction > 0.0) {  // refraction
    float iof = (rt_viewing_ray_payload.inside) ?
                (1.0f / index_of_refraction) : index_of_refraction;
    refract(next_direction, rt_viewing_ray.direction, ffnormal, iof);
    if (rt_viewing_ray_payload.inside) {
      //float p = max(hit_t, 1.0f);
      //rt_viewing_ray_payload.attenuation *= powf(Rho_s.x, p);  // Beer's law, assume Rho_x=y=z
      rt_viewing_ray_payload.attenuation *= Rho_s;
    }
    rt_viewing_ray_payload.inside = !rt_viewing_ray_payload.inside;
  } else {  // specular surface, recursion
    next_direction = reflect(rt_viewing_ray.direction, ffnormal);  // inversed incoming
    //rt_viewing_ray_payload.attenuation *= getSpecularBRDF(reflection_direction,  // incoming
    //                                                      ffnormal,  // normal
    //                                                      -rt_viewing_ray.direction,  // outgoing
    //                                                      Rho_s,  // not Ks but for computing Ks
    //                                                      shininess);  // the power factor
    //rt_viewing_ray_payload.attenuation *= dot(reflection_direction, ffnormal);  // cosine term
    rt_viewing_ray_payload.attenuation *= Rho_s;
  }
  Ray ray(hit_point,
          next_direction,
          rt_viewing_ray_type,
          1e-2f);
  rtTrace(top_object, ray, rt_viewing_ray_payload);
}

// ray tracing, viewing ray, miss, default material
rtDeclareVariable(float3, bg_color, , );  // black

RT_PROGRAM void rt_viewing_ray_miss() {
  HitRecord& hr = hit_record_buffer[launch_index];
  hr.flags = HIT_BACKGROUND;
  hr.attenuation = rt_viewing_ray_payload.attenuation * bg_color;
  hr.position = hr.normal = hr.outgoing = hr.Rho_d = make_float3(0.0f);
}
