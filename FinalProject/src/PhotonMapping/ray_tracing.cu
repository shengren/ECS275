#include "hip/hip_runtime.h"
#include <cfloat>

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"  // tea<>, rnd()
#include "kdtree.h"
#include "structs.h"
#include "inlines.h"

using namespace optix;

// variables used in multiple programs
rtBuffer<float4, 2> output_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, rt_viewing_ray_type, , );
rtDeclareVariable(RTViewingRayPayload, rt_viewing_ray_payload, rtPayload, );

// ray tracing, ray generation
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(float3, camera_position, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float3, camera_w, , );
rtDeclareVariable(uint, sqrt_num_subpixels, , );

RT_PROGRAM void rt_ray_generation() {
  uint seed = tea<16>(launch_index.y * launch_dim.x + launch_index.x,
                      frame_number);
  float2 base = make_float2(launch_index.x * sqrt_num_subpixels,
                            launch_index.y * sqrt_num_subpixels);
  float2 resolution = make_float2(launch_dim.x * sqrt_num_subpixels,
                                  launch_dim.y * sqrt_num_subpixels);
  float3 result = make_float3(0.0f);
  for (int i = 0; i < sqrt_num_subpixels; ++i)
    for (int j = 0; j < sqrt_num_subpixels; ++j) {  // to-do: this anti-aliasing needs a larger stack size
      float2 offset = (base + make_float2(i + rnd(seed), j + rnd(seed)))
                      / resolution * 2.0f - 1.0f;

      Ray ray(camera_position,  // origin
              normalize(offset.x * camera_u + offset.y * camera_v + camera_w),  // direction
              rt_viewing_ray_type,  // type
              1e-2f);  // tmin; tmax uses default

      RTViewingRayPayload payload;
      payload.attenuation = make_float3(1.0f);
      payload.radiance = make_float3(0.0f);
      payload.depth = 1;
      payload.seed = seed;
      payload.inside = false;

      rtTrace(top_object, ray, payload);

      result += payload.radiance;
      seed = payload.seed;
    }
  result *= 1.0f / (float)(sqrt_num_subpixels * sqrt_num_subpixels);

  if (frame_number == 1) {
    output_buffer[launch_index] = make_float4(result, 0.0f);
  } else {
    float a = 1.0f / (float)frame_number;
    float b = ((float)frame_number - 1.0f) * a;
    float3 old_color = make_float3(output_buffer[launch_index]);
    output_buffer[launch_index] = make_float4(a * result + b * old_color, 0.0f);
  }
}

// ray tracing, exception
rtDeclareVariable(float3, bad_color, , );  // blue

RT_PROGRAM void rt_exception() {
  output_buffer[launch_index] = make_float4(bad_color, 0.0f);
  rtPrintExceptionDetails();  // to-do: for debugging
}

// ray tracing, viewing ray, closest hit, default material
rtBuffer<PhotonRecord, 1> photon_map;  // 1D
rtBuffer<ParallelogramLight> lights;  // to-do: only have parallelogram lights
rtDeclareVariable(Ray, rt_viewing_ray, rtCurrentRay, );
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(float3, Rho_d, , );
rtDeclareVariable(float3, Rho_s, , );
rtDeclareVariable(float, shininess, , );  // unused now
rtDeclareVariable(float, index_of_refraction, , );  // non-zero indiates a refraction surface, Rho_s is needed as well
rtDeclareVariable(uint, viewing_ray_max_depth, , );
rtDeclareVariable(uint, rt_shadow_ray_type, , );
rtDeclareVariable(float, radius2, , );

// to-do: photon_map (rtBuffer) cannot be passed through function parameters
// hence this function is not included in 'inlines.h'
// output_buffer is used as a debug buffer in some places
// modified from gather() in progressivePhotonMap/ppm_gather.cu
#define MAX_DEPTH 24  // to-do: 2^24-1 is the maximal size of the photon map

__device__ __inline__ void estimateRadiance(const float3 position,
                                            const float3 normal,
                                            const float3 Rho_d,
                                            const float radius2,
                                            float3& total_flux,
                                            int& num_photons,
                                            float& max_radius2) {
  total_flux = make_float3(0.0f, 0.0f, 0.0f);
  num_photons = 0;  // to-do: unused now
  max_radius2 = 0.0f;

  /*
  const int max_heap_size = (1 << 6) - 1;
  Neighbor max_heap[max_heap_size];
  for (int i = 0; i < max_heap_size; ++i) {
    max_heap[i].dist2 = FLT_MAX;
    max_heap[i].idx = -1;
  }
  */

  unsigned int stack[MAX_DEPTH];
  unsigned int stack_current = 0;
  unsigned int node = 0;  // 0 is the start

#define push_node(N) stack[stack_current++] = (N)
#define pop_node()   stack[--stack_current]

  push_node(0);

  int photon_map_size = photon_map.size();  // for debugging

  do {
    // debugging assertion
    if (!(node < photon_map_size)) {
      //output_buffer[rt_viewing_ray_payload.index] = make_float4(1.0f, 1.0f, 0.0f, 0.0f);
      rtPrintf("overflow case 1\n");
      return;
    }

    const PhotonRecord& pr = photon_map[node];

    if (!(pr.axis & PPM_NULL)) {
      float3 diff = position - pr.position;
      float distance2 = dot(diff, diff);

      // accumulate photons
      if (distance2 <= radius2) {
        //if (dot(normal, pr.normal) > 1e-3f) {  // on the same plane?
        if (dot(normal, pr.incoming) > 1e-3f) {  // to-do: better way?
          total_flux += pr.power * getDiffuseBRDF(Rho_d);  // with BRDF
          num_photons++;
          if (distance2 > max_radius2)
            max_radius2 = distance2;
        }
        /*
        if (dot(normal, pr.normal) > 1e-3f &&
            distance2 < max_heap[0].dist2) {  // heap insertion
          max_heap[0].dist2 = distance2;
          max_heap[0].idx = node;
          int p = 0;
          while (p * 2 + 2 < max_heap_size) {
            if (max_heap[p * 2 + 1].dist2 > max_heap[p * 2 + 2].dist2) {
              if (max_heap[p * 2 + 1].dist2 > max_heap[p].dist2) {
                Neighbor t = max_heap[p * 2 + 1];
                max_heap[p * 2 + 1] = max_heap[p];
                max_heap[p] = t;
                p = p * 2 + 1;
              } else {
                break;
              }
            } else {
              if (max_heap[p * 2 + 2].dist2 > max_heap[p].dist2) {
                Neighbor t = max_heap[p * 2 + 2];
                max_heap[p * 2 + 2] = max_heap[p];
                max_heap[p] = t;
                p = p * 2 + 2;
              } else {
                break;
              }
            }
          }  // while
        }
        */
      }

      // Recurse
      if (!(pr.axis & PPM_LEAF)) {
        float d;
        if (pr.axis & PPM_X) d = diff.x;
        else if (pr.axis & PPM_Y) d = diff.y;
        else d = diff.z;  // PPM_Z

        // Calculate the next child selector. 0 is left, 1 is right.
        int selector = d < 0.0f ? 0 : 1;
        if (d * d < radius2) {
          // debugging assertion
          if (!(stack_current + 1 < MAX_DEPTH)) {
            //output_buffer[rt_viewing_ray_payload.index] = make_float4(1.0f, 1.0f, 0.0f, 0.0f);
            rtPrintf("overflow case 2\n");
            return;
          }

          push_node((node << 1) + 2 - selector);
        }

        // debugging assertion
        if (!(stack_current + 1 < MAX_DEPTH)) {
          //output_buffer[rt_viewing_ray_payload.index] = make_float4(1.0f, 1.0f, 0.0f, 0.0f);
          rtPrintf("overflow case 3\n");
          return;
        }

        node = (node << 1) + 1 + selector;
      } else {
        node = pop_node();
      }
    } else {
      node = pop_node();
    }
  } while (node);

  /*
  for (int i = 0; i < max_heap_size; ++i) {
    if (max_heap[i].idx != -1) {
      total_flux += photon_map[max_heap[i].idx].power * getDiffuseBRDF(Rho_d);  // with BRDF
      num_photons++;
      if (max_heap[i].dist2 > max_radius2)
        max_radius2 = max_heap[i].dist2;
    }
  }
  */
}

// to-do: make this function separate in order to make the code clean
// launch_index, launch_dim, frame_number, lights are local variables
__device__ __inline__ float3 directIllumination(const float3 position,
                                                const float3 normal,
                                                const float3 Rho_d,
                                                uint& seed) {
  // uniformly choose one of the area lights
  int i = (int)((float)lights.size() * rnd(seed));
  const ParallelogramLight& light = lights[i];  // to-do: only one type of light now

  float3 jitter_scale_v1 = light.v1 / (float)light.sqrt_num_samples;
  float3 jitter_scale_v2 = light.v2 / (float)light.sqrt_num_samples;
  int num_samples = light.sqrt_num_samples * light.sqrt_num_samples;
  float ratio = 0.0;
  for (int x = 0; x < light.sqrt_num_samples; ++x)
    for (int y = 0; y < light.sqrt_num_samples; ++y) {
      float3 sample_on_light = light.corner +
                               jitter_scale_v1 * (x + rnd(seed)) +
                               jitter_scale_v2 * (y + rnd(seed));
      float distance_to_light = length(sample_on_light - position);
      float3 direction_to_light = normalize(sample_on_light - position);

      if (dot(normal, direction_to_light) > 1e-2f &&
          dot(light.normal, -direction_to_light) > 1e-2f) {  // trace shadow ray
        RTShadowRayPayload payload;
        payload.blocked = false;

        Ray ray(position,
                direction_to_light,
                rt_shadow_ray_type,
                1e-2f,
                distance_to_light - 1e-2f);

        rtTrace(top_object, ray, payload);  // to-do: hitting the light source doesn't count

        if (!payload.blocked) {
          float geom = getGeometry(normal,
                                   light.normal,
                                   direction_to_light,
                                   distance_to_light);
          ratio += geom;
        }
      }
    }
  ratio *= light.area;  // probability of sampling on this light source
  ratio *= (float)lights.size();  // probability of sampling among light sources
  ratio /= (float)num_samples;

  return light.emitted * getDiffuseBRDF(Rho_d) * ratio;
}

__device__ __inline__ float3 shade(const float3 position,
                                   const float3 normal,
                                   const float3 Rho_d,
                                   uint& seed) {
  // indirect illumination
  float3 total_flux = make_float3(0.0f);
  int num_photons = 0;
  float max_radius2 = 0.0f;

  estimateRadiance(position, normal, Rho_d, radius2, total_flux, num_photons, max_radius2);

  float3 indirect = total_flux / (M_PI * max_radius2);

  // direct illumination
  float3 direct = directIllumination(position, normal, Rho_d, seed);

  float3 ret = direct + indirect;
  //float3 ret = indirect;
  //float3 ret = direct;

  return ret;
}

RT_PROGRAM void rt_viewing_ray_closest_hit() {
  if (fmaxf(Le) > 0.0f) {  // light source?
    rt_viewing_ray_payload.radiance = rt_viewing_ray_payload.attenuation * Le;
    return;
  }

  float3 hit_point = rt_viewing_ray.origin + hit_t * rt_viewing_ray.direction;
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -rt_viewing_ray.direction, world_geometric_normal);

  if (fmaxf(Rho_d) > 0.0f) {  // diffuse surface?
    rt_viewing_ray_payload.radiance = rt_viewing_ray_payload.attenuation *
        shade(hit_point, ffnormal, Rho_d, rt_viewing_ray_payload.seed);
    return;
  }

  if (rt_viewing_ray_payload.depth > viewing_ray_max_depth)
    return;  // stop recursion

  rt_viewing_ray_payload.depth++;

  float3 reflection_direction = reflect(rt_viewing_ray.direction, ffnormal);  // inversed incoming
  float3 refraction_direction;
  float reflection_ratio = 1.0f;
  float refraction_ratio = 0.0f;
  RTViewingRayPayload reflection_payload = rt_viewing_ray_payload;
  RTViewingRayPayload refraction_payload = rt_viewing_ray_payload;
  bool has_refraction = false;
  if (index_of_refraction > 0.0f) {
    float iof = (rt_viewing_ray_payload.inside) ?
                (1.0f / index_of_refraction) : index_of_refraction;
    refract(refraction_direction, rt_viewing_ray.direction, ffnormal, iof);
    float cos_i = dot(-rt_viewing_ray.direction, ffnormal);
    float cos2_t = 1.0f - ((1.0f - (cos_i * cos_i)) / (iof * iof));
    if (cos2_t >= 0) {
      has_refraction = true;
      float a = index_of_refraction - 1.0f;
      float b = index_of_refraction + 1.0f;
      float R0 = a * a / (b * b);
      float c = 1.0f - (rt_viewing_ray_payload.inside ?
                        dot(refraction_direction, -ffnormal) :
                        cos_i);
      reflection_ratio = R0 + (1.0f - R0) * c * c * c * c * c;
      refraction_ratio = 1.0f - reflection_ratio;
    }
  }

  // refraction
  if (has_refraction) {
    refraction_payload.attenuation *= Rho_s * refraction_ratio;
    refraction_payload.inside = !refraction_payload.inside;
    Ray ray(hit_point,
            refraction_direction,
            rt_viewing_ray_type,
            1e-2f);
    rtTrace(top_object, ray, refraction_payload);
    rt_viewing_ray_payload.radiance += refraction_payload.radiance;  // recursively return
    // update the seed
    reflection_payload.seed = refraction_payload.seed;
  }

  // reflection
  reflection_payload.attenuation *= Rho_s * reflection_ratio;  // perfect reflection
  Ray ray(hit_point,
          reflection_direction,
          rt_viewing_ray_type,
          1e-2f);
  rtTrace(top_object, ray, reflection_payload);
  rt_viewing_ray_payload.radiance += reflection_payload.radiance;  // recursively return
  // update the seed
  rt_viewing_ray_payload.seed = reflection_payload.seed;
}

// ray tracing, viewing ray, miss, default material
rtDeclareVariable(float3, bg_color, , );  // black

RT_PROGRAM void rt_viewing_ray_miss() {
  rt_viewing_ray_payload.radiance = rt_viewing_ray_payload.attenuation * bg_color;
}

// ray tracing, direct illumination, shadow ray, any hit
rtDeclareVariable(RTShadowRayPayload, rt_shadow_ray_payload, rtPayload, );

RT_PROGRAM void rt_shadow_ray_any_hit() {
  rt_shadow_ray_payload.blocked = true;
  rtTerminateRay();  // to-do: what's the difference between this function and 'return'
}
