#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "random.h"  // tea<>, rnd
#include "structs.h"

using namespace optix;

// variables used in multiple programs
rtBuffer<HitRecord, 2> hit_record_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, rt_viewing_ray_type, , );
rtDeclareVariable(RTViewingRayPayload, rt_viewing_ray_payload, rtPayload, );

// ray tracing, ray generation
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint, frame_number, , );
rtDeclareVariable(float3, camera_position, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float3, camera_w, , );

RT_PROGRAM void rt_ray_generation() {
  uint seed = tea<16>(launch_index.y * launch_dim.x + launch_index.x,
                              frame_number);
  float2 offset = (make_float2(launch_index)
                  + make_float2(rnd(seed), rnd(seed)))
                  / make_float2(launch_dim) * 2.0f - 1.0f;

  Ray ray(camera_position,  // origin
          normalize(offset.x * camera_u + offset.y * camera_v + camera_w),  // direction
          rt_viewing_ray_type,  // type
          1e-10f);  // tmin; tmax uses default

  RTViewingRayPayload payload;
  payload.attenuation = make_float3(1.0f);
  payload.depth = 0;

  rtTrace(top_object, ray, payload);
}

// ray tracing, exception
rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void rt_exception() {
  HitRecord& hr = hit_record_buffer[launch_index];
  hr.flags = EXCEPTION;
  hr.attenuated_Kd = bad_color;

  rtPrintExceptionDetails();  // to-do:
}

// ray tracing, viewing ray, closest hit, default material
rtDeclareVariable(Ray, rt_viewing_ray, rtCurrentRay, );
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );

RT_PROGRAM void rt_viewing_ray_closest_hit() {
  if (fmaxf(Le) > 0.0f) {  // light source?
    HitRecord& hr = hit_record_buffer[launch_index];
    hr.flags = 0;
    hr.attenuated_Kd = rt_viewing_ray_payload.attenuation * Le;
    return;
  }

  float3 hit_point = rt_viewing_ray.origin + hit_t * rt_viewing_ray.direction;
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -rt_viewing_ray.direction, world_geometric_normal);

  if (fmaxf(Kd) > 0.0f) {  // diffuse surface?
    HitRecord& hr = hit_record_buffer[launch_index];
    hr.flags = HIT;
    hr.attenuated_Kd = rt_viewing_ray_payload.attenuation * Kd;  // to-do: BRDF shouldn't be a constant
    hr.position = hit_point;
    hr.normal = ffnormal;
    hr.outgoing = -rt_viewing_ray.direction;
    return;
  }

  // specular surface, recursion
  rt_viewing_ray_payload.attenuation *= Ks;  // to-do: BRDF shouldn't be a constant
  rt_viewing_ray_payload.depth++;  // to-do: unused
  float3 reflection_direction = reflect(rt_viewing_ray.direction, ffnormal);  // inversed incoming
  Ray ray(hit_point,
          reflection_direction,
          rt_viewing_ray_type,
          1e-10f);
  rtTrace(top_object, ray, rt_viewing_ray_payload);
}

// ray tracing, viewing ray, miss, default material
rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void rt_viewing_ray_miss() {
  HitRecord& hr = hit_record_buffer[launch_index];
  hr.flags = 0;
  hr.attenuated_Kd = rt_viewing_ray_payload.attenuation * bg_color;
}
